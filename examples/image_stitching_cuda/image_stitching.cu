#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <list>
// #include <eigen/Eigen/Core>
#include <eigen/Eigen/Dense>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

// #define BLOCKSIZE 1024
#include "CycleTimer.h"

using Eigen::MatrixXd;


//declare constant memory
__constant__ double homography[9];

__global__ void kernelWarpPerspective(int png_width, int png_height, int curr_width, int curr_height, unsigned char* out_r_device, 
                                    unsigned char* out_g_device, unsigned char* out_b_device, unsigned char* out_a_device, unsigned char* png_r, unsigned char* png_g,
                                    unsigned char* png_b, unsigned char* png_a){
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    if(i > png_height || j > png_width)
        return;

    double prod_00 = homography[0]*j + homography[3]*i + homography[6];
    double prod_10 = homography[1]*j + homography[4]*i + homography[7];
    double prod_20 = homography[2]*j + homography[5]*i + homography[8];
    
    int res_00 = (int)(prod_00/prod_20);
    int res_10 = (int)(prod_10/prod_20);
    if(res_00 >= 0 && res_00 < curr_width && res_10 >= 0 && res_10 < curr_height){
        out_r_device[res_10*curr_width+res_00] = png_r[i*png_width + j]; 
        out_g_device[res_10*curr_width+res_00] = png_g[i*png_width + j]; 
        out_b_device[res_10*curr_width+res_00] = png_b[i*png_width + j];
        out_a_device[res_10*curr_width+res_00] = png_a[i*png_width + j]; 
    }
    
}

void warpPerspective(unsigned char* png_r, unsigned char* png_g, unsigned char* png_b, unsigned char* png_a, 
    int png_width, int png_height, unsigned char* newImR, unsigned char* newImG, unsigned char* newImB, unsigned char* newImA, 
    MatrixXd H, int newIm_width, int newIm_height){
    double overallStartTime = CycleTimer::currentSeconds();
    dim3 blockDim(32, 32, 1);
    dim3 gridDim((png_width + blockDim.x - 1) / blockDim.x, ((png_height + blockDim.y - 1) / blockDim.y));

    // double* H_device;
    double *H_data = H.data();
    // printf("H data %d %d %d %d %d %d %d %d %d", H_data[0], H_data[1], H_data[2], H_data[3], H_data[4], H_data[5], H_data[6], H_data[7], H_data[8]);
    // hipMalloc((void **)&H_device, 3*3*sizeof(double)); //homography
    // hipMemcpy(H_device, H_data, 3*3*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(homography), H_data, sizeof(double)*9);

    unsigned char* png_r_device;
    unsigned char* png_g_device;
    unsigned char* png_b_device;
    unsigned char* png_a_device;
    // hipMalloc((void **)&png_r_device, png_height*png_width*sizeof(char));
    // hipMalloc((void **)&png_g_device, png_height*png_width*sizeof(char));
    // hipMalloc((void **)&png_b_device, png_height*png_width*sizeof(char));
    // hipMalloc((void **)&png_a_device, png_height*png_width*sizeof(char));

    hipMemcpyToSymbol(HIP_SYMBOL(png_r_device), png_r, png_height*png_width*sizeof(char));
    hipMemcpyToSymbol(HIP_SYMBOL(png_g_device), png_g, png_height*png_width*sizeof(char));
    hipMemcpyToSymbol(HIP_SYMBOL(png_b_device), png_b, png_height*png_width*sizeof(char));
    hipMemcpyToSymbol(HIP_SYMBOL(png_a_device), png_a, png_height*png_width*sizeof(char));

    unsigned char* out_r_device;
    unsigned char* out_g_device;
    unsigned char* out_b_device;
    unsigned char* out_a_device;
    hipMalloc((void **)&out_r_device, newIm_width*newIm_height*sizeof(unsigned char)); //try int as well
    hipMalloc((void **)&out_g_device, newIm_width*newIm_height*sizeof(unsigned char));
    hipMalloc((void **)&out_b_device, newIm_width*newIm_height*sizeof(unsigned char));
    hipMalloc((void **)&out_a_device, newIm_width*newIm_height*sizeof(unsigned char));
    
    double startTime = CycleTimer::currentSeconds();
    kernelWarpPerspective<<<gridDim, blockDim>>>(png_width, png_height, newIm_width, newIm_height,
                                                out_r_device, out_g_device, out_b_device, out_a_device, png_r_device, png_g_device,
                                                png_b_device, png_a_device);
    double endTime = CycleTimer::currentSeconds();
    std::cout << "Actual kernel time " << endTime-startTime << std::endl;
    // hipFree(png_r_device);
    // hipFree(png_g_device);
    // hipFree(png_b_device);
    // hipFree(png_a_device);

    
    hipMemcpy(newImR, out_r_device, newIm_width*newIm_height*sizeof(unsigned char), hipMemcpyDeviceToHost); //CHECK ORDER OF ARGS HERE
    hipMemcpy(newImG, out_g_device, newIm_width*newIm_height*sizeof(unsigned char), hipMemcpyDeviceToHost);
    hipMemcpy(newImB, out_b_device, newIm_width*newIm_height*sizeof(unsigned char), hipMemcpyDeviceToHost);
    hipMemcpy(newImA, out_a_device, newIm_width*newIm_height*sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipFree(out_r_device);
    hipFree(out_g_device);
    hipFree(out_b_device);
    hipFree(out_a_device);
    
    double overallEndTime = CycleTimer::currentSeconds();
    std::cout << "Overall warp persp time " << overallEndTime-overallStartTime << std::endl;

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }
}


