#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <list>
// #include <eigen/Eigen/Core>
#include <eigen/Eigen/Dense>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

// #define BLOCKSIZE 1024
#include "CycleTimer.h"

using Eigen::MatrixXd;

double cudaFindPeaks() {
    int *device_input;
    hipMalloc((void **)&device_input, 2 * sizeof(int));
    // int *device_input;
    // int *device_output;
    // int rounded_length = nextPow2(length);
    // hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    // hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    // hipMemcpy(device_input, input, length * sizeof(int), 
    //            hipMemcpyHostToDevice);

    // double startTime = CycleTimer::currentSeconds();
    
    // int result = find_peaks(device_input, length, device_output);

    // hipDeviceSynchronize();
    // double endTime = CycleTimer::currentSeconds();

    // *output_length = result;

    // hipMemcpy(output, device_output, length * sizeof(int),
    //            hipMemcpyDeviceToHost);

    // hipFree(device_input);
    // hipFree(device_output);
    std::cout << "new Running this" << std::endl;

    return 0;
}

// __global__ void kernelInterpolate(MatrixXd* resImg, MatrixXd* copyRes, int start_i, int start_j, double max_x, double max_y){
//     int threadIdxX = blockIdx.x * blockDim.x + threadIdx.x; 
//     int threadIdxY = blockIdx.y * blockDim.y + threadIdx.y; 

//     int i = threadIdxX + start_i;
//     int j = threadIdxY + start_j; 
//     if(i < (int)max_y && j < (int)max_x) {
//         if((*resImg)(i, j) == 0){
//             if (i+1 < max_y && (*copyRes)(i+1,j) != 0){ // && i-1 >=fmax(min_y,0) && j+1 < max_x && j-1 >=fmax(min_x,0) ){
//                 (*resImg)(i, j) = (*copyRes)(i+1,j);
//             }else if(i-1 >= start_i && (*copyRes)(i-1,j) != 0){
//                 (*resImg)(i, j) = (*copyRes)(i-1,j);
//             }else if(j+1 < max_x && (*copyRes)(i,j+1) != 0){
//                 (*resImg)(i, j) = (*copyRes)(i,j+1);
//             }else if(j-1 >= start_j && (*copyRes)(i,j-1) != 0){
//                 (*resImg)(i,j) = (*copyRes)(i,j-1);
//             }else if(i+1 < max_y && j+1 < max_x && (*copyRes)(i+1,j+1)){
//                 (*resImg)(i,j) = (*copyRes)(i+1,j+1);
//             }else if(i-1 >= start_i && j+1 < max_x && (*copyRes)(i-1,j+1)){
//                 (*resImg)(i,j) = (*copyRes)(i-1,j+1);
//             }else if(i+1 < max_y && j-1 >= start_j && (*copyRes)(i+1,j-1)){
//                 (*resImg)(i,j) = (*copyRes)(i+1,j-1);
//             }else if(i-1 >= start_i && j-1 >=start_j && (*copyRes)(i-1,j-1)){
//                 (*resImg)(i,j) = (*copyRes)(i-1,j-1);
//             }
//         }
//     }
// }

// __global__ void kernelCompose(MatrixXd *resImg, MatrixXd *newImage, int start_i, int start_j, double max_x, double max_y){
//     int threadIdxX = blockIdx.x * blockDim.x + threadIdx.x; 
//     int threadIdxY = blockIdx.y * blockDim.y + threadIdx.y; 

//     int i = threadIdxX + start_i;
//     int j = threadIdxY + start_j; 
//     if(i < (int)max_y && j < (int)max_x) {
//         if ((*resImg)(i,j) == 0){
//             (*resImg)(i,j) = (*newImage)(i,j);
//         }
//         if ((*resImg)(i,j) != 0 && (*newImage)(i, j) != 0){
//             (*resImg)(i,j) = fmax((*newImage)(i,j), (*resImg)(i,j));
//         }
//     }
// }

// void placeImage(MatrixXd newImage, MatrixXd* resImg, double min_x, double min_y, double max_x, double max_y){
//     // int w = newImage.cols();
//     // int h = newImage.rows();
//     // printf("w: %d, h: %d", w, h);
//     int start_i = (int)fmax(min_y,0);
//     int start_j = (int)fmax(min_x,0);
//     MatrixXd *resImg_device; 
//     MatrixXd *newImage_device; 

//     std::cout << "Size of matrixxd " << sizeof(MatrixXd) << std::endl;
//     hipMalloc((void **)&resImg_device, sizeof(double)*sizeof(resImg->rows()*resImg->cols()));
//     hipMalloc((void **)&newImage_device, sizeof(double)*sizeof(newImage.rows()*newImage.cols()));
//     hipMemcpy(resImg_device, resImg, sizeof(double)*sizeof(resImg->rows()*resImg->cols()), hipMemcpyHostToDevice);
//     hipMemcpy(newImage_device, &newImage, sizeof(double)*sizeof(newImage.rows()*newImage.cols()), hipMemcpyHostToDevice);
    
//     dim3 blockDim(16, 16, 1);
//     dim3 gridDim((((int)max_y) - start_i + blockDim.x - 1) / blockDim.x, (((int)max_x) - start_j + blockDim.y - 1) / blockDim.y);
//     kernelCompose<<<gridDim, blockDim>>>(resImg_device, newImage_device, start_i, start_j, max_x, max_y); 
//     hipMemcpy(resImg, resImg_device, sizeof(double)*sizeof(resImg->rows()*resImg->cols()), hipMemcpyDeviceToHost);
//     // std::cout << *resImg << std::endl;
//     // #pragma omp parallel for schedule(dynamic)
//     // for (int i = start_i; i < (int)max_y; i++){ //access as row col
//     //     for (int j = start_j; j < (int)max_x; j++){
//     //         if ((*resImg)(i,j) == 0){
//     //             (*resImg)(i,j) = newImage(i,j);
//     //         }
//     //         if ((*resImg)(i,j) != 0 && newImage(i, j) != 0){
//     //             (*resImg)(i,j) = fmax(newImage(i,j), (*resImg)(i,j));
//     //         }
//     //     }
//     // }
    
    
//     MatrixXd copyRes = (*resImg);
//     // dim3 blockDim(16, 16, 1);
//     // dim3 gridDim((((int)max_y) - start_i + blockDim.x - 1) / blockDim.x, (((int)max_x) - start_j + blockDim.y - 1) / blockDim.y);
//     // MatrixXd *resImg_device; 
//     // MatrixXd *copyRes_device; 

//     // hipMalloc((void **)&resImg_device, sizeof(MatrixXd));
//     // hipMalloc((void **)&copyRes_device, sizeof(MatrixXd));
//     // hipMemcpy(resImg_device, resImg, sizeof(MatrixXd), hipMemcpyHostToDevice);
//     // hipMemcpy(copyRes_device, &copyRes, sizeof(MatrixXd), hipMemcpyHostToDevice);
    
//     // kernelInterpolate<<<gridDim, blockDim>>>(resImg_device, copyRes_device, start_i, start_j, max_x, max_y);

//     // hipMemcpy(resImg, resImg_device, sizeof(MatrixXd), hipMemcpyDeviceToHost);

//     #pragma omp parallel for schedule(dynamic)
//     for(int i = start_i; i < (int)max_y; i++){
//         for(int j = start_j; j < (int)max_x; j++){
//             if((*resImg)(i, j) == 0){
//                 if (i+1 < max_y && copyRes(i+1,j) != 0){ // && i-1 >=fmax(min_y,0) && j+1 < max_x && j-1 >=fmax(min_x,0) ){
//                     (*resImg)(i, j) = copyRes(i+1,j);
//                 }else if(i-1 >= fmax(min_y,0) && copyRes(i-1,j) != 0){
//                     (*resImg)(i, j) = copyRes(i-1,j);
//                 }else if(j+1 < max_x && copyRes(i,j+1) != 0){
//                     (*resImg)(i, j) = copyRes(i,j+1);
//                 }else if(j-1 >=fmax(min_x,0) && copyRes(i,j-1) != 0){
//                     (*resImg)(i,j) = copyRes(i,j-1);
//                 }else if(i+1 < max_y && j+1 < max_x && copyRes(i+1,j+1)){
//                     (*resImg)(i,j) = copyRes(i+1,j+1);
//                 }else if(i-1 >= fmax(min_y,0) && j+1 < max_x && copyRes(i-1,j+1)){
//                     (*resImg)(i,j) = copyRes(i-1,j+1);
//                 }else if(i+1 < max_y && j-1 >=fmax(min_x,0) && copyRes(i+1,j-1)){
//                     (*resImg)(i,j) = copyRes(i+1,j-1);
//                 }else if(i-1 >= fmax(min_y,0) && j-1 >=fmax(min_x,0) && copyRes(i-1,j-1)){
//                     (*resImg)(i,j) = copyRes(i-1,j-1);
//                 }
//             }
//         }
//     }
// }
__global__ void kernelWarpPerspective(double* H, int png_width, int png_height, int newImCols, int newImRows, unsigned char* out_r_device, 
                                    unsigned char* out_g_device, unsigned char* out_b_device, unsigned char* out_a_device, unsigned char* png_r, unsigned char* png_g,
                                    unsigned char* png_b, unsigned char* png_a){
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if(i > png_height && j > png_width)
        return;
    
    int tmp00 = j;
    int tmp10 = i;
    int tmp20 = 1;

    double prod_00 = H[0]*tmp00 + H[1]*tmp10 + H[2]*tmp20;
    double prod_10 = H[3]*tmp00 + H[4]*tmp10 + H[5]*tmp20;
    double prod_20 = H[6]*tmp00 + H[7]*tmp10 + H[8]*tmp20;
    
    int res_00 = (int)prod_00/prod_20;
    int res_10 = (int)prod_10/prod_20;
    if(res_00 >= 0 && res_00 < newImCols && res_10 >= 0 && res_10 < newImRows){
        out_r_device[res_10*newImCols+res_10] = (int)png_r[i*png_width + j]; //try flipped too
        out_g_device[res_10*newImCols+res_10] = (int)png_g[i*png_width + j]; //try flipped too
        out_b_device[res_10*newImCols+res_10] = (int)png_b[i*png_width + j]; //try flipped too
        out_a_device[res_10*newImCols+res_10] = (int)png_a[i*png_width + j]; //try flipped too
    }
    
}

void warpPerspective(unsigned char* png_r, unsigned char* png_g, unsigned char* png_b, unsigned char* png_a, 
int png_width, int png_height, MatrixXd* newImR,MatrixXd* newImG,MatrixXd* newImB, MatrixXd* newImA, MatrixXd H){
    dim3 blockDim(16, 16, 1);
    dim3 gridDim((png_width + blockDim.x - 1) / blockDim.x, ((png_height + blockDim.y - 1) / blockDim.y));

    double* H_device;
    double *H_data = H.data();
    hipMalloc((void **)&H_device, 3*3*sizeof(double)); //homography
    hipMemcpy(H_device, H_data, 3*3*sizeof(double), hipMemcpyHostToDevice);

    unsigned char* png_r_device;
    unsigned char* png_g_device;
    unsigned char* png_b_device;
    unsigned char* png_a_device;
    hipMalloc((void **)&png_r_device, png_height*png_width*sizeof(char));
    hipMalloc((void **)&png_g_device, png_height*png_width*sizeof(char));
    hipMalloc((void **)&png_b_device, png_height*png_width*sizeof(char));
    hipMalloc((void **)&png_a_device, png_height*png_width*sizeof(char));

    hipMemcpy(png_r_device, png_r, png_height*png_width*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(png_g_device, png_g, png_height*png_width*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(png_b_device, png_b, png_height*png_width*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(png_a_device, png_a, png_height*png_width*sizeof(char), hipMemcpyHostToDevice);

    unsigned char* out_r_device;
    unsigned char* out_g_device;
    unsigned char* out_b_device;
    unsigned char* out_a_device;
    hipMalloc((void **)&out_r_device, newImR->rows()*newImR->cols()*sizeof(unsigned char)); //try int as well
    hipMalloc((void **)&out_g_device, newImG->rows()*newImG->cols()*sizeof(unsigned char));
    hipMalloc((void **)&out_b_device, newImB->rows()*newImB->cols()*sizeof(unsigned char));
    hipMalloc((void **)&out_a_device, newImA->rows()*newImA->cols()*sizeof(unsigned char));
    
    kernelWarpPerspective<<<gridDim, blockDim>>>(H_device, png_width, png_height, newImR->cols(), newImG->rows(), 
                                                out_r_device, out_g_device, out_b_device, out_a_device, png_r_device, png_g_device,
                                                png_b_device, png_a_device);

    hipFree(H_device);
    hipFree(png_r_device);
    hipFree(png_g_device);
    hipFree(png_b_device);
    hipFree(png_a_device);

    //May not have to malloc here
    unsigned char* out_r_host = (unsigned char*)malloc(newImR->rows()*newImR->cols()*sizeof(unsigned char));
    unsigned char* out_g_host = (unsigned char*)malloc(newImR->rows()*newImR->cols()*sizeof(unsigned char));
    unsigned char* out_b_host = (unsigned char*)malloc(newImR->rows()*newImR->cols()*sizeof(unsigned char));
    unsigned char* out_a_host = (unsigned char*)malloc(newImR->rows()*newImR->cols()*sizeof(unsigned char));

    hipMemcpy(out_r_device, out_r_host, png_height*png_width*sizeof(char), hipMemcpyDeviceToHost); //CHECK ORDER OF ARGS HERE
    hipMemcpy(out_g_device, out_g_host, png_height*png_width*sizeof(char), hipMemcpyDeviceToHost);
    hipMemcpy(out_b_device, out_b_host, png_height*png_width*sizeof(char), hipMemcpyDeviceToHost);
    hipMemcpy(out_a_device, out_a_host, png_height*png_width*sizeof(char), hipMemcpyDeviceToHost);

    *newImR = Eigen::Map<MatrixXd>(out_r_device);
    *newImG = Eigen::Map<MatrixXd>(out_g_device);
    *newImB = Eigen::Map<MatrixXd>(out_b_device);
    *newImA = Eigen::Map<MatrixXd>(out_a_device);

    hipFree(out_r_device);
    hipFree(out_g_device);
    hipFree(out_b_device);
    hipFree(out_a_device);
    free(out_r_host);
    free(out_g_host);
    free(out_b_host);
    free(out_a_host);


    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }
}

// void warpPerspective(unsigned char* png_r, unsigned char* png_g, unsigned char* png_b, unsigned char* png_a, 
//         int png_width, int png_height, MatrixXd* newImR,MatrixXd* newImG,MatrixXd* newImB, MatrixXd* newImA, MatrixXd H){
//     //FIX: Need to create matrix of form Nx3 and do the matrix multiply all at once- cuda kernel
//     int i; 
//     #pragma omp parallel for collapse(2)
//     for(i=0; i< png_height; i++){ 
//         for(int j=0; j<png_width; j++){
//             MatrixXd tmp = MatrixXd::Constant(3,1, 0.0);
//             tmp(0,0) = j;
//             tmp(1,0) = i;
//             tmp(2,0) = 1;
//             MatrixXd res = H*tmp;
//             MatrixXd tm =  Matslice(res, 2, 0, 1, res.cols()).replicate(3,1); //(MatrixXd array, int start_row, int start_col, int height, int width)
//             res = res.cwiseQuotient(tm);
//             if ((int)res(0,0) >= 0 && (int)res(0,0) < (*newImR).cols() && (int)res(1,0) >= 0 && (int)res(1,0) < (*newImR).rows()){
//                 (*newImR)((int)res(1,0), (int)res(0,0)) = (int)png_r[i*png_width + j];
//                 (*newImG)((int)res(1,0), (int)res(0,0)) = (int)png_g[i*png_width + j];
//                 (*newImB)((int)res(1,0), (int)res(0,0)) = (int)png_b[i*png_width + j];
//                 (*newImA)((int)res(1,0), (int)res(0,0)) = (int)png_a[i*png_width + j];
//             }
//         }
//     }
// }

