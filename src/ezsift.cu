#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "ezsift.h"
#include "common.h"
#include "vvector.h"
#include "image.h"
#include "CycleTimer.h"

namespace ezsift {

void printCudaInfo();

std::vector<std::vector<std::vector<Image<float>>>> pyramids_gpu(std::vector<std::vector<Image<unsigned char>>> all_octaves, 
                                                    std::vector<int> all_nOctaves, 
                                                    int nGpyrLayers, int nDogLayers, int nLayers) {
    
    hipDeviceSynchronize();
    
    // std::cout << "Noctaves " << nOctaves << " nGpyrLayers " << nGpyrLayers  << std::endl;
    // std::cout << "Running this from ezsift" << std::endl;

    return 0;
}

void printCudaInfo()
{
    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}

} // end namespace ezsift